#include "hip/hip_runtime.h"
#include "./World.h"
#include "./Log.hpp"
#include "./Util.hpp"

#include <exception>
#include <cmath>
#include <utility>
#include <algorithm>

#include "math.h"

// ====================== Definitions =======================

Data data;

// ======================== Kernels =========================

#define tid (threadIdx)
#define bid (blockIdx)
#define dim (blockDim)

struct KernelParticle
{
    Point location;
    Point velocity;
};

const int MaxParticleCount = 4096;
__constant__ KernelParticle kernelParticleSrc[MaxParticleCount];

__global__ void GForceEffect(
    float dt, int size,
    KernelParticle* src, KernelParticle* dst,
    float attrC, float attrP, float replC, float replP,
    float maxF, float mass, bool assign)
{
    int cid = tid.x + bid.x * dim.x;
    Point c = src[cid].location;
    
    Point acc = Point{0.0f, 0.0f};
    
    for(int i=0; i<size; i++)
    {
        if(i == cid) continue;
        Point p = kernelParticleSrc[i].location;
        Point dir = p(c);
        float dist = dir.len();
        acc = acc + (1.0f / dist) * (replC / pow(dist, replP) - attrC / pow(dist, attrP)) * (1.0f / mass) * dir;
    }
    
    if(assign)
    {
        dst[cid].location = src[cid].velocity * dt + 0.5f * acc * dt * dt;
        dst[cid].velocity = acc * dt;
    }
    else
    {
        dst[cid].location = dst[cid].location + src[cid].velocity * dt + 0.5f * acc * dt * dt;
        dst[cid].velocity = dst[cid].velocity + acc * dt;
    }
}

// ========================= Tests ==========================

// ======================= Simulation =======================

void Step(float dt)
{
    const float delay = GetParameter("GeneratingDelay");
    const float mass = GetParameter("Mass");
    const float replCoinc = GetParameter("RepulsionCoincident");
    const float replLevel = GetParameter("RepulsionLevel");
    const float attrCoinc = GetParameter("AttractionCoincident");
    const float attrLevel = GetParameter("AttractionLevel");
    const float limitCoinc = GetParameter("LimitCoincident");
    const float limitLevel = GetParameter("LimitLevel");
    const float maxForce = GetParameter("MaxForce");
    
    // generate.
    if(delay >= 1e-4f)
    {
        float usedTime = 0;
        data.freeTime += dt;
        while(data.freeTime >= 0.0f)
        {
            data.freeTime -= delay;
            usedTime += delay;
            data.particles.emplace_back(Particle {
                data.genFrom + RandReal() * (data.genTo - data.genFrom)
                    + data.initVelocity * usedTime,
                data.initVelocity,
                RandInt() % data.selective == 0 });
        }
    }
    
    // move.
    {
        const int n = data.particles.size();
        
        HostArray<KernelParticle> hdata(n);
        for(int i=0; i<n; i++)
        {
            hdata[i].location = data.particles[i].location;
            hdata[i].velocity = data.particles[i].velocity;
        }
        
        KernelArray<KernelParticle> kdata(hdata);
        KernelArray<KernelParticle> gdata(hdata);
        
        for(int base = 0; base < n; base += MaxParticleCount)
        {
            int m = min(MaxParticleCount, n - base) * sizeof(KernelParticle);
            hipMemcpyToSymbol(HIP_SYMBOL(kernelParticleSrc), hdata.pointer + base,m);
            const int threadCount = 256;
            GForceEffect<<<n / threadCount + 1, threadCount>>>(
                dt, m,
                kdata.pointer, gdata.pointer,
                attrCoinc, attrLevel, replCoinc, replLevel,
                maxForce, mass, base == 0);
        }
        
        while(true)
        {
            hipError_t err = hipGetLastError();
            if(hipSuccess != err) Log("CUDA error: %s\n", hipGetErrorString(err));
            else break;
        }
        
        hdata <<= gdata;
        for(int i=0; i<n; i++)
        {
            data.particles[i].location = hdata[i].location;
            data.particles[i].velocity = hdata[i].velocity;
        }
    }
    
    // remove.
    {
        data.particles.erase(
            std::remove_if(
                data.particles.begin(),
                data.particles.end(),
                [](Particle const& p)
                {
                    return p.location.x < data.limitLB.x
                        || data.limitRT.x < p.location.x
                        || p.location.y < data.limitLB.y
                        || data.limitRT.y < p.location.y;
                }),
            data.particles.end());
    }
    
    // re-arrange array.
    {
        // move shown particles to the left side of array.
        auto const& pos = std::partition(data.particles.begin(), data.particles.end(), [](Particle const& p)
        {
            return p.shown;
        });
        
        // assign value here so that the count needs no re-calculation.
        data.shownCount = (int)(pos - data.particles.begin());
    }
}

#pragma region InterfaceFunctions

void SetParameter(const char* s, float value)
{
    if(data.parameters.find(s) == data.parameters.end())
    {
        Log("Dynamically create new parameter: %s\n", s);
    }
    data.parameters[s] = value;
}

float GetParameter(const char* s)
{
    if(data.parameters.find(s) != data.parameters.end())
    {
        return data.parameters[s];
    }
    Log("Cannot find value of: %s\n", s);
    return NAN;
}

void EnvInit()
{
    data = Data();
    // Physics parameters.
    data.parameters["Mass"] = 1.0f;
    data.parameters["RepulsionCoincident"] = 1.0f;
    data.parameters["RepulsionLevel"] = 3.0f;
    data.parameters["AttractionCoincident"] = 1.0f;
    data.parameters["AttractionLevel"] = 2.0f;
    data.parameters["MaxForce"] = 100.0f;
    data.parameters["LimitCoincident"] = 1.0f;
    data.parameters["LimitLevel"] = 1.0f;
    // System parameters.
    data.parameters["GeneratingDelay"] = 0.05f;
    // System states.
    data.shownCount = 0;
    data.freeTime = 0;
}

void EnvDispose()
{
    
}

int GetMeshVertexCount()
{
    return data.mesh.size();
}

Vector2* GetMesh()
{
    if(data.mesh.size() == 0) return nullptr;
    data.vec2Output.clear();
    for(int i=0; i<data.mesh.size(); i++)
    {
        data.vec2Output.push_back(Vector2 {
            data.mesh[i].x,
            data.mesh[i].y });
    }
    return &data.vec2Output[0];
}

void SetMesh(Vector2* arr, int count)
{
    data.mesh.clear();
    for(int i=0; i<count; i++)
    {
        Point v;
        v.x = arr[i].x;
        v.y = arr[i].y;
        data.mesh.push_back(v);
    }
}

void SetSelectiveConstant(int x)
{
    if(x < 1) return;
    data.selective = x;
}

int GetSelectiveConstant()
{
    return data.selective;
}


int GetParticlesCount(bool selective)
{
    return selective ? data.shownCount : data.particles.size();
}

Vector2* GetParticles(bool selective)
{
    data.vec2Output.clear();
    for(int i = 0; i < data.shownCount; i++)
    {
        data.vec2Output.emplace_back(Vector2 {
            data.particles[i].location.x,
            data.particles[i].location.y });
    }
    return &data.vec2Output[0];
}

void RemoveAllParticles()
{
    data.particles.clear();
}

void SetInitialVelocity(Vector2 t)
{
    data.initVelocity.x = t.x;
    data.initVelocity.y = t.y;
}

Vector2 GetInitialVelocity()
{
    Vector2 val;
    val.x = data.initVelocity.x;
    val.y = data.initVelocity.y;
    return val;
}

void SetGeneratingLine(Rectangle area)
{
    data.genFrom.x = area.x;
    data.genFrom.y = area.y;
    data.genTo.x = area.x + area.w;
    data.genTo.y = area.y + area.h;
}

Rectangle GetGeneratingLine()
{
    Rectangle area;
    area.x = data.genFrom.x;
    area.y = data.genFrom.y;
    area.w = data.genTo.x - data.genFrom.x;
    area.h = data.genTo.y - data.genFrom.y;
    return area;
}

void SetLimitArea(Rectangle area)
{
    data.limitLB.x = area.x;
    data.limitLB.y = area.y;
    data.limitRT.x = area.x + area.w;
    data.limitRT.y = area.y + area.h;
}

Rectangle GetLimitArea()
{
    Rectangle area;
    area.x = data.limitLB.x;
    area.y = data.limitLB.y;
    area.w = data.limitRT.x - data.limitLB.x;
    area.h = data.limitRT.y - data.limitLB.y;
    return area;
}

#pragma endregion InterfaceFunctions
