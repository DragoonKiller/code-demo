#include "hip/hip_runtime.h"

// ====================================================================================================================
// ====================================================================================================================
// ====================================================================================================================

template<typename T>
__global__ void Add(T* a, T* b, T* c)
{
    c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

void TestSumOfArray()
{
    const int N = 1000;
    HostArray<int> A(N);
    HostArray<int> B(N);
    HostArray<int> C(N);
    KernelArray<int> KA(N);
    KernelArray<int> KB(N);
    KernelArray<int> KC(N);
    for(int i=0; i<N; i++) A[i] = i + 1;
    for(int i=0; i<N; i++) B[i] = A[i] * A[i];
    KA <<= A;
    KB <<= B;
    Add<<<N, 1>>>(KA.pointer, KB.pointer, KC.pointer);
    C <<= KC;
    for(int i=0; i<N; i++) Log("%d + %d = %d\n", A[i], B[i], C[i]);
}

// ====================================================================================================================
// ====================================================================================================================
// ====================================================================================================================


template<typename T>
__global__ void PrefixSum(T* s, T* a)
{
    int tid = threadIdx.x;
    
    s[tid] = 0;
    for(int i=0; i<5; i++)
    {
        s[tid] += a[(tid + i) % blockDim.x];
    }
}

void TestThreadAccess()
{
    const int N = 1000;
    HostArray<int> A(N);
    HostArray<int> B(N);
    KernelArray<int> KA(N);
    KernelArray<int> KB(N);
    for(int i=0; i<N; i++) A[i] = i + 2;
    KA <<= A;
    PrefixSum<<<1, N>>>(KB.pointer, KA.pointer);
    B <<= KB;
    for(int i=0; i<N; i++) Log("sum of 5 elements: %d : %d\n", A[i], B[i]);
}
